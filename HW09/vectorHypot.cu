/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin College by Joel C. Adams.
 * Extended further by Quentin Barnes, for HW09, Calvin University, Nov 2019
 * Changed to compute the Hypotenus.
 */

/**
 * Vector hypot: C = sqrt( A*A + B*B).
 *
 * This sample is a very basic sample that implements takeing the 
 * hypotenus from two elements. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <omp.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the hypotenus given A and B into C. 
 * The 3 vectors have the same number of elements numElements.
 */
__global__
void vectorHypot(const float *A, const float *B, float *C, unsigned long numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = sqrt( A[i]*A[i] + B[i]*B[i] );
    }
}

void checkErr(hipError_t err, const char* msg) 
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{
    double runtimeCopyTo, runtimeCompute, runtimeCopyBack, runtimeSeq;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 50000;
    if (argc == 2) {
      numElements = strtoul( argv[1] , 0, 10 );
    }
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %lu elements]\n", numElements);

    // Allocate the host input vectors A & B
    float * h_A = (float *)malloc(size);
    float * h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float * h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // 1a. Allocate the device input vectors A & B
    float * d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    checkErr(err, "Failed to allocate device vector A");
    float * d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    checkErr(err, "Failed to allocate device vector B");

    // 1.b. Allocate the device output vector C
    float * d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    checkErr(err, "Failed to allocate device vector C");

    runtimeCopyTo = -omp_get_wtime();

    // 2. Copy the host input vectors A and B in host memory 
    //     to the device input vectors in device memory
    // printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector A from host to device");


    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector B from host to device");

    runtimeCopyTo += omp_get_wtime();
    runtimeCompute = -omp_get_wtime();

    // 3. Launch the Vector Hypot CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorHypot<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorHypot kernel");

    runtimeCompute += omp_get_wtime();
    runtimeCopyBack = -omp_get_wtime();

    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy vector C from device to host");

    runtimeCopyBack += omp_get_wtime();

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrt(h_A[i]*h_A[i] + h_B[i]*h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("CUDA test PASSED\n");
//    printf("CUDA time: %lf\n", stopTime-startTime); 

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_B);
    checkErr(err, "Failed to free device vector B");

    err = hipFree(d_C);
    checkErr(err, "Failed to free device vector C");

    runtimeSeq = -omp_get_wtime();
    // repeat the computation sequentially
    for (int i = 0; i < numElements; ++i)
    {
       h_C[i] = sqrt(h_A[i]*h_A[i] + h_B[i]*h_B[i]);
    }
    runtimeSeq += omp_get_wtime();

    // verify again
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrt(h_A[i]*h_A[i] + h_B[i]*h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("\nNormal test PASSED\n");
//    printf("Normal time: %lf\n", stopTime-startTime); 

    //Print times
    printf("Time spent (Cuda): \n\tcopying the A and B arrays: \t%f\n\tcomputing the sum: \t\t%f\n\tcopying the C array: \t\t%f\n\tTotal: \t\t\t\t%f\n", runtimeCopyTo, runtimeCopyBack, runtimeCompute, (runtimeCopyTo + runtimeCopyBack + runtimeCompute));
    printf("Time spent (Sequential): \t\t%f\n", runtimeSeq);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("Done\n");
    return 0;
}

